#include "hip/hip_runtime.h"
/**
* @file      rasterize.cu
* @brief     CUDA-accelerated rasterization pipeline.
* @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
* @date      2012-2016
* @copyright University of Pennsylvania & STUDENT
*/

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "registrationTools.h"
#include "registration.h"
#include "kdtree.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#include <thrust/execution_policy.h>
#include <util/svd3.h>

#ifndef imax
#define imax(a, b) (((a) > (b)) ? (a) : (b))
#endif

#ifndef imin
#define imin(a, b) (((a) < (b)) ? (a) : (b))
#endif

#define EXHAUSTIVE 1
#define KDTREE 1

#define blockSize 128
#define scene_scale 100.0f
#define threadsPerBlock(blockSize)

template<typename T>
__host__ __device__

void swap(T &a, T &b) {
	T tmp(a);
	a = b;
	b = tmp;
}

static int numObjects;

static glm::vec3 *dev_pos_fixed = NULL;
static glm::vec3 *dev_pos_rotated = NULL;
static glm::vec3 *dev_pos_corr = NULL;
static glm::vec3 *dev_pos_rotated_centered = NULL;
static glm::mat3 *dev_w = NULL;
static Node *dev_kd = NULL;

//static hipEvent_t start, stop;
/**
* Kernel that writes the image to the OpenGL PBO directly.
*/
/******************
* copyPtsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, int offset, glm::vec3 *pos, float *vbo) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / scene_scale;

	if (index < N) {
		vbo[4 * (index + offset) + 0] = pos[index].x * c_scale;
		vbo[4 * (index + offset) + 1] = pos[index].y * c_scale;
		vbo[4 * (index + offset) + 2] = pos[index].z * c_scale;
		vbo[4 * (index + offset) + 3] = 1.0f;
	}
}

__global__ void kernCopyColorsToVBO(int N, int offset, glm::vec3 color, float *vbo) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * (index + offset) + 0] = color.x;
		vbo[4 * (index + offset) + 1] = color.y;
		vbo[4 * (index + offset) + 2] = color.z;
		vbo[4 * (index + offset) + 3] = 1.0f;
	}
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void copyPointsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, 0, dev_pos_fixed, vbodptr_positions);
	checkCUDAError("copyPositionsFixed failed!");

	kernCopyColorsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, 0, glm::vec3(1.0f, 1.0f, 1.0f),
		vbodptr_velocities);
	checkCUDAError("copyColorsFixed failed!");

	kernCopyPositionsToVBO << < fullBlocksPerGrid, blockSize >> >(numObjects, numObjects,
		dev_pos_rotated, vbodptr_positions);
	checkCUDAError("copyPositionsRotated failed!");

	kernCopyColorsToVBO << < fullBlocksPerGrid, blockSize >> >(numObjects, numObjects, glm::vec3(0.3f, 0.9f, 0.3f),
		vbodptr_velocities);
	checkCUDAError("copyColorsRotated failed!");


	hipDeviceSynchronize();
}


__global__ void kernInitializePosArray(int N, glm::vec3 *pos, float scale) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		pos[index].x *= scale;
		pos[index].y *= scale;
		pos[index].z *= scale;
	}
}


__global__ void transformPoints(int N, glm::vec3 *pos_in, glm::vec3 *pos_out, glm::mat4 transformation) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		pos_out[index] = glm::vec3(transformation * glm::vec4(pos_in[index], 1.0f));
	}
}


glm::mat4 constructTransformationMatrix(const glm::vec3 &translation, const glm::vec3& rotation, const glm::vec3& scale) {
	glm::mat4 translation_matrix = glm::translate(glm::mat4(), translation);
	glm::mat4 rotation_matrix = glm::rotate(glm::mat4(), rotation.x, glm::vec3(1, 0, 0));
	rotation_matrix *= glm::rotate(glm::mat4(), rotation.y, glm::vec3(0, 1, 0));
	rotation_matrix *= glm::rotate(glm::mat4(), rotation.z, glm::vec3(0, 0, 1));
	glm::mat4 scale_matrix = glm::scale(glm::mat4(), scale);
	return translation_matrix* rotation_matrix * scale_matrix;
}

glm::mat4 constructTranslationMatrix(const glm::vec3 &translation) {
	glm::mat4 translation_matrix = glm::translate(glm::mat4(), translation);
	return translation_matrix;
}

/**
* Called once at the beginning of the program to allocate memory.
*/
void registrationInitGPU(const std::vector<glm::vec3>& pts) {
	numObjects = (int)pts.size();
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	hipMalloc((void**)&dev_pos_fixed, numObjects * sizeof(glm::vec3));
	hipMalloc((void**)&dev_pos_rotated, numObjects * sizeof(glm::vec3));
	hipMalloc((void**)&dev_pos_corr, numObjects * sizeof(glm::vec3));
	hipMalloc((void**)&dev_pos_rotated_centered, numObjects * sizeof(glm::vec3));
	hipMalloc((void**)&dev_w, numObjects * sizeof(glm::mat3));

	checkCUDAError("registration Init");

	hipMemcpy(dev_pos_fixed, &pts[0], numObjects * sizeof(glm::vec3), hipMemcpyHostToDevice);
	checkCUDAError("pos_fixed Memcpy");

#if KDTREE
	hipMalloc((void**)&dev_kd, numObjects * sizeof(Node));
	KDTree(pts);
	std::vector<Node> tree = pts.getTree();
	hipMemcpy(dev_kd, &tree[0], numObjects * sizeof(Node), hipMemcpyHostToDevice);
#endif

	kernInitializePosArray << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_fixed, scene_scale);

	glm::mat4 transformation = constructTransformationMatrix(glm::vec3(1.0f, 0.0f, 0.0f),
		glm::vec3(0.4f, 0.4f, -0.2f), glm::vec3(1.0f, 1.0f, 1.0f));

	transformPoints << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_fixed,
		dev_pos_rotated, transformation);
}


__global__ void findNearestNeighborExhaustive(int N, const glm::vec3 *source, const glm::vec3 *target, glm::vec3 *corr) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		glm::vec3 pt = source[index];
		float d_closest = glm::distance(target[0], pt);
		int i = 0;
		for (int j = 1; j < N; j++) {
			float d = glm::distance(target[j], pt);
			if (d < d_closest) {
				d_closest = d;
				i = j;
			}
		}
		corr[index] = target[i];
	}
}


__device__ float calculateHyperPlaneDist(const glm::vec3& pt1, const glm::vec3& pt2, int axis){
	if (axis == 0) return pt1.x - pt2.x;
	else if (axis == 1) return pt1.y - pt2.y;
	else return pt1.z - pt2.z;
}


__global__ void findNearestNeighborKDTree(int N, const glm::vec3 *source, const Node *tree, glm::vec3 *corr){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(index < N{
		glm::vec3 pt = source[index];
		float d_closest = glm::distance(tree[0].data, pt);
		bool explored = false;
		float hyper_dist = calculateHyperPlaneDist(pt, tree[0].data, tree[0].axis);
		int curr_node = hyper_dist < 0 ? tree[0].left, tree[0].right;
		int closest_node = 0;
		while(1){
			// explore current node & below
			while(curr_node != -1){
				float d = glm::distance(tree[curr_node].data, pt);
				if (d < d_closest){
					d_closest = d;
					closest_node = curr_node;
				}
				hyper_dist = calculateHyperPlaneDist(pt, tree[curr_node].data, tree[curr_node].axis);
				curr_node = hyper_dist < 0 ? tree[curr_node].left, tree[curr_node].right;

			}
			if(explored) break;
			else{
				int parent = tree[closest_node].parent;
				if (parent == -1) break;
				hyper_dist = calculateHyperPlaneDist(pt, tree[parent].data, tree[parent].axis);
				if (abs(hyper_dist) < d_closest){
					curr_node = hyper_dist < 0 ? tree[parent].eft, tree[parent].right;
				}else break;
			}


		}
		corr[index] = tree[closest_node].data;
	}
}


__global__ void translatePts(int N, glm::vec3* pos_in, glm::vec3* pos_out, glm::mat4 translation) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		pos_out[index] = glm::vec3(translation * glm::vec4(pos_in[index], 1.f));
	}
}


__global__ void calculateW(int N, glm::vec3* pos_rotated, glm::vec3* pos_cor, glm::mat3* w) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		w[index] = glm::outerProduct(pos_rotated[index], pos_cor[index]);
		// w[index] = glm::mat3(pos_rotated[index] * pos_cor[index].x,
			// pos_rotated[index] * pos_cor[index].y, pos_rotated[index] * pos_cor[index].z);
	}
}


// __global__ void calculateSVDWrapper(glm::mat3& w, glm::mat3& S, glm::mat3& U, glm::mat3 &V) {
// 	svd(w[0][0], w[0][1], w[0][2], w[1][0], w[1][1], w[1][2], w[2][0], w[2][1], w[2][2],
// 		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
// 		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
// 		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]);
// }




/**
* Perform point cloud registration.
*/
void registration() {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
#if EXHAUSTIVE
	findNearestNeighborExhaustive << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_rotated,
		dev_pos_fixed, dev_pos_corr);
#elif KDTREE
	findNearestNeighborKDTree << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_rotated,
		dev_kd , dev_pos_corr);
#endif

	checkCUDAError("Find nearest Neighbor");

	// ICP Algorithm based on http://ais.informatik.uni-freiburg.de/teaching/ss11/robotics/slides/17-icp.pdf

	// calculate mean of two point clouds using stream compaction
	thrust::device_ptr<glm::vec3> thrust_pos_corr(dev_pos_corr);
	thrust::device_ptr<glm::vec3> thrust_pos_rotated(dev_pos_rotated);

	glm::vec3 pos_corr_mean = thrust::reduce(thrust_pos_corr, thrust_pos_corr + numObjects,
		glm::vec3(0.f, 0.f, 0.f));
	pos_corr_mean /= numObjects;

	glm::vec3 pos_rotated_mean = glm::vec3(thrust::reduce(thrust_pos_rotated,
		thrust_pos_rotated + numObjects, glm::vec3(0.f, 0.f, 0.f))) ;
	pos_rotated_mean /= numObjects;

    glm::mat4 translation_matrix = constructTranslationMatrix(-pos_corr_mean);
	translatePts << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_corr, dev_pos_corr, translation_matrix);
		
    translation_matrix = constructTranslationMatrix(-pos_rotated_mean);
	translatePts << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_rotated, dev_pos_rotated_centered, translation_matrix);
		
	checkCUDAError("Translating Pts");

	calculateW << < fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_rotated_centered, dev_pos_corr, dev_w);
	thrust::device_ptr<glm::mat3> thrust_w(dev_w);
	glm::mat3 W = thrust::reduce(thrust_w, thrust_w + numObjects, glm::mat3(0.f));
	checkCUDAError("Calculated W");

	glm::mat3 S, U, V;

    // could not get gpu version working
	// calculateSVDWrapper <<<1, 1 >>> (W, S, U, V);
	// checkCUDAError("SVD W");
    // the faster implementation creates error that makes det non-one
    svd(W[0][0], W[0][1], W[0][2], W[1][0], W[1][1], W[1][2], W[2][0], W[2][1], W[2][2],
     U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
     S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
     V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]);

	glm::mat3 R = glm::transpose(U) * V;
	float det_R = glm::determinant(R);
	//glm::mat4 scale_matrix = glm::scale(glm::mat4(), glm::vec3(1.f / det_R, 1.f / det_R, 1.f / det_R));
	glm::vec3 t = pos_corr_mean - R * pos_rotated_mean;
	glm::mat4 T = glm::translate(glm::mat4(), t);
	//glm::mat4 transformation = T * glm::mat4(R) * scale_matrix;
    glm::mat4 transformation = T * glm::mat4(R);
	float det_transform = glm::determinant(transformation);
	transformPoints << < fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos_rotated, dev_pos_rotated, transformation);
	checkCUDAError("Transforming Pts");
}

/**
* Called once at the end of the program to free CUDA memory.
*/
void registrationFree() {

	// deconstruct primitives attribute/indices device buffer

	hipFree(dev_pos_rotated);
	hipFree(dev_pos_fixed);
	hipFree(dev_pos_rotated_centered);
	hipFree(dev_pos_corr);
	hipFree(dev_w);
	cudaFrree(dev_kd);

	dev_pos_fixed = NULL;
	dev_pos_rotated = NULL;
	dev_pos_corr = NULL;
	dev_pos_rotated_centered = NULL;
	dev_w = NULL;

	checkCUDAError("registration Free");
}


template <typename T>
T calculate_vector_mean(std::vector<T> input){
	T sum;
	for (auto &element: input){
		sum += element;
	}
	return sum /= input.size();
};


std::vector<glm::vec3> registration_init_cpu(std::vector<glm::vec3> &input){
    glm::mat4 transformation = constructTransformationMatrix(glm::vec3(1.0f, 0.0f, 0.0f),
        glm::vec3(0.4f, 0.4f, -0.2f), glm::vec3(1.0f, 1.0f, 1.0f));
    std::vector<glm::vec3>result(input.size(), glm::vec3(0.f, 0.f, 0.f));
    for (int i = 0; i < input.size(); ++i){
        result[i] = glm::vec3(transformation * glm::vec4(input[i], 1.0f));
    }
	return result;
}

// skeleton code for cpu_step; no display, just for performance comparison
void registration_cpu(std::vector<glm::vec3>& target, std::vector<glm::vec3>& source){
	int numPts = target.size();

	std::vector<glm::vec3> corr(numPts, glm::vec3(0.f, 0.f, 0.f));

	for (int k = 0; k < numPts; k++){
		auto best_dist = glm::distance(source[k], target[0]);
		int i = 0;
		for (int j = 1; j < numPts; j++){
			auto d = glm::distance(source[k], target[j]);
			if ( d < best_dist){
				best_dist = d;
				i = j;
			}
		}
		corr[k] = target[i];
	}


	glm::vec3 mean_corr = calculate_vector_mean(corr);
	glm::vec3 mean_source = calculate_vector_mean(source);

	std::vector<glm::vec3> source_centered = source;

	for (int i = 0; i < numPts; i++){
		source_centered[i] = source[i] - mean_source;
		corr[i] -= mean_corr;
	}

	// calculate w
	std::vector<glm::mat3> w(numPts, glm::mat3(glm::vec3(0.f, 0.f, 0.f), glm::vec3(0.f, 0.f, 0.f), glm::vec3(0.f, 0.f, 0.f)));
	for (int i = 0; i < numPts; i++){
		w[i] = glm::outerProduct(source_centered[i], corr[i]);
	}

	glm::mat3 W = calculate_vector_mean(w);
	W *= numPts;

	glm::mat3 S, U, V;

	svd(W[0][0], W[0][1], W[0][2], W[1][0], W[1][1], W[1][2], W[2][0], W[2][1], W[2][2],
		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]);

	glm::mat3 R = glm::transpose(U) * V;
	glm::vec3 t = mean_corr - R * mean_source;
	glm::mat4 T = glm::translate(glm::mat4(), t);
	glm::mat4 transformation = T * glm::mat4(R);

	for (int i = 0; i < numPts; i++){
		source[i] = glm::vec3(transformation * glm::vec4(source[i], 1.0f));
	}

}